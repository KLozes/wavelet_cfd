#include "hip/hip_runtime.h"

#include <stdio.h>
#include "MultiLevelSparseGridKernels.cuh"

__global__ void initGridKernel(MultiLevelSparseGrid &grid) {
  // initialize the blocks of the base grid level
  i32 i = threadIdx.x + blockIdx.x*blockDim.x;
	i32 j = threadIdx.y + blockIdx.x*blockDim.y;
  i32 k = threadIdx.z + blockIdx.x*blockDim.z;
  if (i < grid.baseGridSize[0]/blockSize && 
      j < grid.baseGridSize[1]/blockSize && 
      k < grid.baseGridSize[2]/blockSize) {
    grid.activateBlock(0, i, j, k);
  }
}

__global__ void updateIndicesKernel(MultiLevelSparseGrid &grid) {
  // update the hashtable with new sorted indices
  START_BLOCK_LOOP

    if (grid.bLocList[bIdx] != kEmpty) {
      grid.bIdxList[bIdx] = bIdx;
      grid.hashTable.insertValue(grid.bLocList[bIdx], bIdx);
    }

  END_BLOCK_LOOP
}

__global__ void updatePrntIndicesKernel(MultiLevelSparseGrid &grid) {
  // update the parent indices list
  START_BLOCK_LOOP

    i32 lvl, ib, jb, kb;
    u64 loc = grid.bLocList[bIdx];
    grid.decode(loc, lvl, ib, jb, kb);

    if (lvl > 0) {
      u64 pLoc = grid.encode(lvl-1, ib/2, jb/2, kb/2);
      u32 prntIdx = grid.hashTable.getValue(pLoc);  
      grid.prntIdxList[bIdx] = prntIdx;
    }

  END_BLOCK_LOOP
}


__global__ void updateNbrIndicesKernel(MultiLevelSparseGrid &grid) {

  START_BLOCK_LOOP

    i32 lvl, ib, jb, kb;
    u64 loc = grid.bLocList[bIdx];
    grid.decode(loc, lvl, ib, jb, kb);

    u32 idx = 0;
    for (i32 dk=-1; dk<2; dk++) {
      for(int dj=-1; dj<2; dj++) {
        for(int di=-1; di<2; di++) {
          u64 nbrLoc = grid.encode(lvl, ib+di, jb+dj, kb+dk);
          grid.nbrIdxList[bIdx*27+idx] = grid.hashTable.getValue(nbrLoc);
          idx++;
        }
      }
    }

  END_BLOCK_LOOP

}

__global__ void flagActiveCellsKernel(MultiLevelSparseGrid &grid) {

  START_CELL_LOOP

    i32 lvl, ib, jb, kb;
    u64 loc = grid.bLocList[bIdx];
    grid.decode(loc, lvl, ib, jb, kb);

    if (grid.isInteriorBlock(lvl, ib, jb, kb)) {

      u32 idx000 = grid.getNbrIdx(bIdx, i-haloSize, j-haloSize, k-haloSize);
      u32 idx100 = grid.getNbrIdx(bIdx, i+haloSize, j-haloSize, k-haloSize);
      u32 idx010 = grid.getNbrIdx(bIdx, i-haloSize, j+haloSize, k-haloSize);
      u32 idx110 = grid.getNbrIdx(bIdx, i+haloSize, j+haloSize, k-haloSize);
      u32 idx001 = grid.getNbrIdx(bIdx, i-haloSize, j-haloSize, k+haloSize);
      u32 idx101 = grid.getNbrIdx(bIdx, i+haloSize, j-haloSize, k+haloSize);
      u32 idx011 = grid.getNbrIdx(bIdx, i-haloSize, j+haloSize, k+haloSize);
      u32 idx111 = grid.getNbrIdx(bIdx, i+haloSize, j+haloSize, k+haloSize);

      u32 cEmpty = bEmpty * blockSizeTot;
      grid.cFlagsList[cIdx] = ACTIVE;
      if (idx000 >= cEmpty || idx100 >= cEmpty || idx010 >= cEmpty || idx110 >= cEmpty ||
          idx001 >= cEmpty || idx101 >= cEmpty || idx011 >= cEmpty || idx111 >= cEmpty) {
        grid.cFlagsList[cIdx] = GHOST;
      }

    }

  END_CELL_LOOP
}

__global__ void flagParentCellsKernel(MultiLevelSparseGrid &grid) {

  START_CELL_LOOP

    i32 lvl, ib, jb, kb;
    u64 loc = grid.bLocList[bIdx];
    grid.decode(loc, lvl, ib, jb, kb);

    i32 cFlag = grid.cFlagsList[cIdx];

    if (lvl > 0 && grid.isInteriorBlock(lvl, ib, jb, kb) && (cFlag == ACTIVE || cFlag == PARENT)) {

      // parent block memory index
      u32 prntIdx = grid.prntIdxList[bIdx];

      // parent cell local indices
      i32 ip = i/2 + ib%2 * blockSize / 2;
      i32 jp = j/2 + jb%2 * blockSize / 2;
      i32 kp = k/2 + kb%2 * blockSize / 2;

      // parent cell memory index
      u32 pIdx = grid.getNbrIdx(prntIdx, ip, jp, kp);

      grid.cFlagsList[pIdx] = PARENT;

    }

  END_CELL_LOOP
}

__global__ void addFineBlocksKernel(MultiLevelSparseGrid &grid) {

  START_BLOCK_LOOP

    i32 lvl, ib, jb, kb;
    u64 loc = grid.bLocList[bIdx];
    grid.decode(loc, lvl, ib, jb, kb);

    if (grid.isInteriorBlock(lvl, ib, jb, kb)) {
      if (lvl == 0 || grid.bFlagsList[bIdx] == REFINE) {
        // add finer blocks if not already on finest level
        grid.bFlagsList[bIdx] = KEEP;
        if (lvl < grid.nLvls-1) {
          for (i32 dk=0; dk<=1; dk++) {
            for (i32 dj=0; dj<=1; dj++) {
              for (i32 di=0; di<=1; di++) {
                grid.activateBlock(lvl+1, 2*ib+di, 2*jb+dj, 2*kb+dk);
              }
            }
          }
        }
      } 
    }

  END_BLOCK_LOOP

}

__global__ void setBlocksKeepKernel(MultiLevelSparseGrid &grid) {

  START_BLOCK_LOOP

    if (grid.bFlagsList[bIdx] == NEW ) {
      grid.bFlagsList[bIdx] = KEEP;
    }

  END_BLOCK_LOOP
}

__global__ void setBlocksDeleteKernel(MultiLevelSparseGrid &grid) {

  START_BLOCK_LOOP

    grid.bFlagsList[bIdx] = DELETE;

  END_BLOCK_LOOP
}

__global__ void addAdjacentBlocksKernel(MultiLevelSparseGrid &grid) {

  START_BLOCK_LOOP

    i32 lvl, ib, jb, kb;
    u64 loc = grid.bLocList[bIdx];
    grid.decode(loc, lvl, ib, jb, kb);

    if (grid.isInteriorBlock(lvl, ib, jb, kb) && grid.bFlagsList[bIdx] == KEEP) {
      // add neighboring blocks
      for (i32 dk=-1; dk<=1; dk++) {
        for (i32 dj=-1; dj<=1; dj++) {
          for (i32 di=-1; di<=1; di++) {
            grid.activateBlock(lvl, ib+di, jb+dj, kb+dk);
          }
        }
      }
    }

  END_BLOCK_LOOP
}

__global__ void addReconstructionBlocksKernel(MultiLevelSparseGrid &grid) {

  START_BLOCK_LOOP

    // activate parents and neghbors needed for wavelet transform
    i32 lvl, ib, jb, kb;
    u64 loc = grid.bLocList[bIdx];
    grid.decode(loc, lvl, ib, jb, kb);

    if (grid.isInteriorBlock(lvl, ib, jb, kb) && lvl > 2 && grid.bFlagsList[bIdx] == KEEP) {
      for (i32 dk=-1; dk<=1; dk++) {
        for (i32 dj=-1; dj<=1; dj++) {
          for (i32 di=-1; di<=1; di++) {
            grid.activateBlock(lvl-1, ib/2+di, jb/2+dj, kb/2+dk);
          }
        }
      }
    }

  END_BLOCK_LOOP
}

__global__ void deleteDataKernel(MultiLevelSparseGrid &grid) {

  START_CELL_LOOP

    if (grid.bFlagsList[bIdx] == DELETE) {
      if (cIdx % blockSizeTot == 0) {
        grid.bLocList[bIdx] = kEmpty;
        grid.bIdxList[bIdx] = bEmpty;
        atomicAdd(&(grid.nBlocks), -1);
      }
      grid.cFlagsList[cIdx] = 0;
      for(i32 f=0; f<grid.nFields; f++) {
        real *F = grid.getField(f);
        F[cIdx] = 0;
      }
    }

  END_CELL_LOOP
}

__global__ void addBoundaryBlocksKernel(MultiLevelSparseGrid &grid) {

  START_BLOCK_LOOP

    i32 lvl, ib, jb, kb;
    u64 loc = grid.bLocList[bIdx];
    grid.decode(loc, lvl, ib, jb, kb);

    if (grid.isInteriorBlock(lvl, ib, jb, kb) && 
       (ib == 0 || ib == grid.baseGridSize[0]/blockSize*powi(2,lvl)-1 ||
        jb == 0 || jb == grid.baseGridSize[1]/blockSize*powi(2,lvl)-1 ||
        kb == 0 || kb == grid.baseGridSize[2]/blockSize*powi(2,lvl)-1)) {
      // add neighboring exterior blocks
      for (i32 dk=-1; dk<=1; dk++) {
        for (i32 dj=-1; dj<=1; dj++) {
          for (i32 di=-1; di<=1; di++) {
            if (grid.isExteriorBlock(lvl, ib+di, jb+dj, kb+dk)) {
              grid.activateBlock(lvl, ib+di, jb+dj, kb+dk);            
            }
          }
        }
      }
    }

  END_BLOCK_LOOP
}

__global__ void computeImageDataKernel(MultiLevelSparseGrid &grid, i32 f) {

  bool gridOn = true;

  real *U;
  if (f >= 0) {
    U = grid.getField(f);
  }

  START_CELL_LOOP

    u64 loc = grid.bLocList[bIdx];
    i32 lvl, ib, jb, kb;
    grid.decode(loc, lvl, ib, jb, kb);

    if (grid.isInteriorBlock(lvl, ib, jb, kb) && loc != kEmpty && grid.cFlagsList[cIdx] == ACTIVE) {
      u32 nPixels = powi(2,(grid.nLvls - 1 - lvl));
      for (uint jj=0; jj<nPixels; jj++) {
        for (uint ii=0; ii<nPixels; ii++) {
          u32 iPxl = ib*blockSize*nPixels + i*nPixels + ii;
          u32 jPxl = jb*blockSize*nPixels + j*nPixels + jj;
          if (f >= 0) {
            grid.imageData[jPxl*grid.imageSize[0] + iPxl] = U[cIdx];
          }
          else {
            grid.imageData[jPxl*grid.imageSize[0] + iPxl] = (lvl+1);
          }
          if (f < 0 && gridOn && ii > 0 && jj > 0) {
            grid.imageData[jPxl*grid.imageSize[0] + iPxl] = 0;
          }
        }
      }
    }

  END_CELL_LOOP
}