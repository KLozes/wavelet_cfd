#include <string.h>
#include <iostream>
#include <ctime>

#include "CompressibleSolver.cuh"

int main(int argc, char* argv[]) {
  dataType domainSize[2] = {1.0, 1.0};
  u32 baseGridSize[2] = {blockSize*10, blockSize*10};
  u32 nLvls = 6;
  dataType cfl = .8;
  dataType waveletThresh = .01;
  dataType tStep = .005;

  CompressibleSolver *solver = new CompressibleSolver(domainSize, baseGridSize, nLvls, cfl, waveletThresh);
  solver->icType = 0;
  solver->bcType = 0;
  solver->immerserdBcType = 0;
  solver->initialize();

  dataType t = 0;
  i32 n = 0;
  while(t < 100) {

    t += solver->step(tStep);
    n += 1;

    solver->paint();
    printf("n: %d, t = %f\n", n, t);

  }
  
  hipDeviceSynchronize();
  delete solver;
	hipDeviceReset();
}
