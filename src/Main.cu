#include <string.h>
#include <iostream>
#include <ctime>

#include "CompressibleSolver.cuh"

int main(int argc, char* argv[]) {
  dataType domainSize[2] = {1.0, 2.0};
  u32 baseGridSize[2] = {blockSize*100, blockSize*2*100};
  u32 nLvls = 1;

  CompressibleSolver *solver = new CompressibleSolver(domainSize, baseGridSize, nLvls, .3);
  solver->initializeBaseGrid();
  solver->setInitialConditions(0);
  solver->setBoundaryConditions(0);
  solver->paint();

  dataType t = 0;
  i32 n = 0;
  while(t < 100) {

    if (n % 1 == 0) {
      solver->computeDeltaT();
    }

    for (i32 stage = 0; stage<3; stage++) {
      solver->computeRightHandSide();
      solver->updateFields(stage);
      solver->setBoundaryConditions(0);
    }
    hipDeviceSynchronize();
    t += solver->deltaT;
    n++;

    if (n % 50 == 0) {
      printf("n: %d, t = %f\n", n, t);
      solver->paint();
    }

  }

  hipDeviceSynchronize();
  delete solver;
	hipDeviceReset();
}
