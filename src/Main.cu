#include <string.h>
#include <iostream>
#include <ctime>

#include "CompressibleSolver.cuh"

int main(int argc, char* argv[]) {
  dataType domainSize[2] = {1.0, 1.0};
  u32 baseGridSize[2] = {blockSize*100, blockSize*100};
  u32 nLvls = 1;

  CompressibleSolver *solver = new CompressibleSolver(domainSize, baseGridSize, nLvls);
  solver->initializeBaseGrid();
  solver->setInitialConditions(0);
  solver->setBoundaryConditions(0);
  solver->paint();

  dataType t = 0;
  i32 n = 0;
  while(t < 100) {

    solver->computeDeltaT();

    for (i32 stage = 0; stage<3; stage++) {
      solver->computeRightHandSide();
      solver->updateFields(stage);
      solver->setBoundaryConditions(0);
    }
    hipDeviceSynchronize();
    t += solver->deltaT;
    n++;

    if (n % 100 == 0) {
      printf("n: %d, t = %f\n", n, t);
      solver->paint();
    }

  }

  hipDeviceSynchronize();
  delete solver;
	hipDeviceReset();
}
