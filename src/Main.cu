#include <string.h>
#include <iostream>
#include <chrono>

#include "CompressibleSolver.cuh"

int main(int argc, char* argv[]) {
  real domainSize[2] = {1.0, 1.0};
  u32 baseGridSize[2] = {blockSize*10, blockSize*10};
  u32 nLvls = 8;
  real cfl = .80;
  real waveletThresh = .005;
  real tStep = .02;

  CompressibleSolver *solver = new CompressibleSolver(domainSize, baseGridSize, nLvls, cfl, waveletThresh);
  solver->icType = 0;
  solver->bcType = 0;
  solver->immerserdBcType = 0;
  solver->initialize();

  real t = 0;
  i32 n = 0;
  while(t < 100) {

    t += solver->step(tStep);
    n += 1;

    solver->paint();
    printf("n: %d, t = %f, tSolver = %d, tGrid = %d, nBlocks = %d\n", n, t, solver->tSolver , solver->tGrid, solver->hashTable.nKeys);

  }
  
  hipDeviceSynchronize();
  delete solver;
	hipDeviceReset();
}
