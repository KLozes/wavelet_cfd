#include <string.h>
#include <iostream>
#include <ctime>

#include "CompressibleSolver.cuh"

int main(int argc, char* argv[]) {
  dataType domainSize[2] = {4.0, 1.0};
  u32 baseGridSize[2] = {blockSize*100, blockSize*25};
  u32 nLvls = 4;
  dataType cfl = .1;
  dataType waveletThresh = .004;
  dataType tStep = .02;

  CompressibleSolver *solver = new CompressibleSolver(domainSize, baseGridSize, nLvls, cfl, waveletThresh);
  solver->icType = 0;
  solver->bcType = 0;
  solver->immerserdBcType = 0;
  solver->initialize();

  dataType t = 0;
  i32 n = 0;
  while(t < 100) {

    t += solver->step(tStep);
    n += 1;

    solver->paint();
    printf("n: %d, t = %f\n", n, t);

  }
  
  hipDeviceSynchronize();
  delete solver;
	hipDeviceReset();
}
