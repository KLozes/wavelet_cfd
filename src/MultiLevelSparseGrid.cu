#include "hip/hip_runtime.h"
#include <thrust/sort.h>
#include <algorithm>

#include <png++/png.hpp>
#include "MultiLevelSparseGrid.cuh"
#include "MultiLevelSparseGridKernels.cuh"

MultiLevelSparseGrid::MultiLevelSparseGrid(dataType *domainSize_, u32 *baseGridSize_, u32 nLvls_, u32 nFields_) {

  domainSize[0] = domainSize_[0];
  domainSize[1] = domainSize_[1];

  baseGridSize[0] = baseGridSize_[0];
  baseGridSize[1] = baseGridSize_[1];

  nLvls = nLvls_;
  nFields = nFields_;

  imageSize[0] = (baseGridSize[0])*powi(2,nLvls-1);  // image size is the max resolution not including boundary condition blocks
  imageSize[1] = (baseGridSize[1])*powi(2,nLvls-1);

  blockCounter = 0;
  imageCounter = 0;


  // grid size checking
  assert(isPowerOf2(blockSize));
  assert(baseGridSize[0]*baseGridSize[1]/blockSize/blockSize < nBlocksMax);

  hipMallocManaged(&bLocList, nBlocksMax*sizeof(u64));
  hipMallocManaged(&bIdxList, nBlocksMax*sizeof(u32));
  hipMallocManaged(&bFlagsList, nBlocksMax*sizeof(u32));
  hipMallocManaged(&prntIdxList, nBlocksMax*sizeof(u32));
  hipMallocManaged(&nbrIdxList, blockHaloSizeTot*nBlocksMax*sizeof(u32));
  hipMallocManaged(&cFlagsList, blockSizeTot*nBlocksMax*sizeof(u32));
  hipMallocManaged(&fieldData, nFields*blockSizeTot*nBlocksMax*sizeof(dataType));
  hipMallocManaged(&imageData, blockSizeTot*nBlocksMax*sizeof(dataType));

  hipMemset(bLocList, 0, nBlocksMax*sizeof(u64));
  hipMemset(bIdxList, 0, nBlocksMax*sizeof(u32));
  hipMemset(bFlagsList, 0, nBlocksMax*sizeof(u32));
  hipMemset(prntIdxList, 0, nBlocksMax*sizeof(u32));
  hipMemset(nbrIdxList, 0, blockHaloSizeTot*nBlocksMax*sizeof(u32));
  hipMemset(cFlagsList, 0, blockSizeTot*nBlocksMax*sizeof(u32));
  hipMemset(fieldData, 0, nFields*blockSizeTot*nBlocksMax*sizeof(dataType));
  hipMemset(imageData, 0, blockSizeTot*nBlocksMax*sizeof(dataType));

  hipDeviceSynchronize();
}

MultiLevelSparseGrid::~MultiLevelSparseGrid(void) {
  hipDeviceSynchronize();
  hipFree(bLocList);
  hipFree(bIdxList);
  hipFree(prntIdxList);
  hipFree(nbrIdxList);
  hipFree(cFlagsList);
  hipFree(fieldData);
  hipFree(imageData);
}

void MultiLevelSparseGrid::initializeBaseGrid(void) {
  // fill the bLocList with base grid blocks
  initGridKernel<<<nBlocksMax/cudaBlockSize+1, cudaBlockSize>>>(*this);
  hipDeviceSynchronize();
  nBlocks = hashTable.nKeys;
  
  addBoundaryBlocksKernel<<<nBlocks/cudaBlockSize+1, cudaBlockSize>>>(*this);
  hipDeviceSynchronize();
  nBlocks = hashTable.nKeys;

  // sort the data by location code
  sortBlocks();
  hipDeviceSynchronize();
}

void MultiLevelSparseGrid::adaptGrid(void) {

  addFineBlocksKernel<<<nBlocks/cudaBlockSize+1, cudaBlockSize>>>(*this);
  hipDeviceSynchronize();
  nBlocks = hashTable.nKeys;

  addAdjacentBlocksKernel<<<nBlocks/cudaBlockSize+1, cudaBlockSize>>>(*this);
  hipDeviceSynchronize();
  nBlocks = hashTable.nKeys;

  addReconstructionBlocksKernel<<<nBlocks/cudaBlockSize+1, cudaBlockSize>>>(*this);
  hipDeviceSynchronize();
  nBlocks = hashTable.nKeys;

  addBoundaryBlocksKernel<<<nBlocks/cudaBlockSize+1, cudaBlockSize>>>(*this);
  hipDeviceSynchronize();
  nBlocks = hashTable.nKeys;

  deleteDataKernel<<<nBlocks*blockSizeTot/cudaBlockSize+1, cudaBlockSize>>>(*this);
  hipDeviceSynchronize();

  updatePrntIndicesKernel<<<nBlocks/cudaBlockSize+1, cudaBlockSize>>>(*this);
}

void MultiLevelSparseGrid::sortBlocks(void) {

  thrust::sort_by_key(thrust::device, bLocList, bLocList+nBlocks, bIdxList);
  sortFieldData();
  hipDeviceSynchronize();

  hashTable.reset();
  hipDeviceSynchronize();
  updateIndicesKernel<<<nBlocks/cudaBlockSize+1, cudaBlockSize>>>(*this);
  hipDeviceSynchronize();
  nBlocks = hashTable.nKeys;

  updatePrntIndicesKernel<<<nBlocks/cudaBlockSize+1, cudaBlockSize>>>(*this);
  updateNbrIndicesKernel<<<nBlocks*blockHaloSizeTot/cudaBlockSize+1, cudaBlockSize>>>(*this);
  updateCellFlagsKernel<<<nBlocks*blockSizeTot/cudaBlockSize+1, cudaBlockSize>>>(*this);
  hipDeviceSynchronize();

  /*
  for (u32 bIdx = 0; bIdx<nBlocks; bIdx++) {
    u64 loc = bLocList[bIdx];
    i32 lvl, ib, jb;
    mortonDecode(loc, lvl, ib, jb);

    printf("%d %d\n", ib, jb);
    for (i32 j=blockHaloSize-1; j>=0; j--) {
      for (i32 i=0; i<blockHaloSize; i++) {
        u32 idx = nbrIdxList[bIdx*blockHaloSizeTot + j*blockHaloSize + i];
        printf("%8d ", idx);
      }
      printf("\n");
    }
    printf("\n");
  }
  */
  
}

__host__ __device__ void MultiLevelSparseGrid::getCellPos(i32 lvl, i32 ib, i32 jb, i32 i, i32 j, dataType *pos) {
  pos[0] = (ib*blockSize + i + .5)*getDx(lvl);
  pos[1] = (jb*blockSize + j + .5)*getDy(lvl);
}

__host__ __device__ u32 MultiLevelSparseGrid::getNbrIdx(u32 bIdx, i32 i, i32 j) {
  return nbrIdxList[bIdx*blockHaloSizeTot + (j+haloSize)*blockHaloSize + (i+haloSize)];
}

__host__ __device__ dataType MultiLevelSparseGrid::getDx(i32 lvl) {
  return dataType(domainSize[0])/dataType(baseGridSize[0]*powi(2,lvl));
}

__host__ __device__ dataType MultiLevelSparseGrid::getDy(i32 lvl) {
  return dataType(domainSize[1])/dataType(baseGridSize[1]*powi(2,lvl));
}

__host__ __device__ bool MultiLevelSparseGrid::isInteriorBlock(i32 lvl, i32 i, i32 j) { 
  i32 gridSize[2] = {i32(baseGridSize[0]/blockSize*powi(2,lvl)), 
                     i32(baseGridSize[1]/blockSize*powi(2,lvl))};
  return i >= 0 && j >= 0 && i < gridSize[0] && j < gridSize[1];
}

__host__ __device__ bool MultiLevelSparseGrid::isExteriorBlock(i32 lvl, i32 i, i32 j) {
  return !isInteriorBlock(lvl, i, j);
}

__host__ __device__ dataType* MultiLevelSparseGrid::getField(u32 f) {
  return &fieldData[f*nBlocksMax*blockSizeTot];
}

__host__ __device__ void MultiLevelSparseGrid::activateBlock(i32 lvl, i32 i, i32 j) {
  u64 loc = mortonEncode(lvl, i, j);
  u32 idx = hashTable.insert(loc);

  if (idx != bEmpty) { 
    // new key was inserted if not bEmpty
    bLocList[idx] = loc;
    bIdxList[idx] = idx;
    bFlagsList[idx] = KEEP;
  }

}

// seperate bits from a given integer 3 positions apart
__host__ __device__ u64 MultiLevelSparseGrid::split(u32 a) {
  u64 x = (u64)a & ((1<<20)-1); // we only look at the first 20 bits
  x = (x | x << 32) & 0x1f00000000ffff;
  x = (x | x << 16) & 0x1f0000ff0000ff;
  x = (x | x << 8) & 0x100f00f00f00f00f;
  x = (x | x << 4) & 0x10c30c30c30c30c3;
  x = (x | x << 2) & 0x1249249249249249;
  return x;
}

// encode ijk indices and resolution level into morton code
__host__ __device__ u64 MultiLevelSparseGrid::mortonEncode(i32 lvl, i32 i, i32 j) {
  u64 morton = 0;
  i += 1; // add one so that boundary blocks are no longer negative negative
  j += 1;
  morton |= (u64)lvl << 60 | split(i) | split(j) << 1;
  return morton;
}

// compact separated bits into into an integer
__host__ __device__ u32 MultiLevelSparseGrid::compact(u64 w) {
  w &=                  0x1249249249249249;
  w = (w ^ (w >> 2))  & 0x30c30c30c30c30c3;
  w = (w ^ (w >> 4))  & 0xf00f00f00f00f00f;
  w = (w ^ (w >> 8))  & 0x00ff0000ff0000ff;
  w = (w ^ (w >> 16)) & 0x00ff00000000ffff;
  w = (w ^ (w >> 32)) & 0x00000000001fffff;
  return (u32)w;
}

// decode morton code into ij idx and resolution level
__host__ __device__ void MultiLevelSparseGrid::mortonDecode(u64 morton, i32 &lvl, i32 &i, i32 &j) {
  lvl = i32((morton & ((u64)15 << 60)) >> 60);   // get the level stored in the last 4 bits
  morton &= ~ ((u64)15 << 60); // remove the last 4 bits
  i = compact(morton) - 1; 
  j = compact(morton >> 1) - 1;
}

void MultiLevelSparseGrid::paint(void) {

  hipDeviceSynchronize();
  png::image<png::gray_pixel_16> image(imageSize[0], imageSize[1]);

  bool drawGrid = false;

  for (i32 f=-1; f<4; f++) {
    computeImageData(f);

    // find the field maximum and minimum of the image field
    dataType maxVal = -1e32;
    dataType minVal = 1e32;

    for (u32 bIdx = 0; bIdx < nBlocks; bIdx++) {
      u64 loc = bLocList[bIdx];
      i32 lvl, ib, jb;
      mortonDecode(loc, lvl, ib, jb);
      if (isInteriorBlock(lvl, ib, jb) && loc != kEmpty) {
        for (u32 idx = 0; idx < blockSizeTot; idx++) {
          dataType val = imageData[bIdx*blockSizeTot + idx];
          maxVal = max(maxVal, val);
          minVal = min(minVal, val);
        }
      }
    }

    // normalize the image field data
    for (u32 bIdx = 0; bIdx < nBlocks; bIdx++) {
      u64 loc = bLocList[bIdx];
      i32 lvl, ib, jb;
      mortonDecode(loc, lvl, ib, jb);
      if (isInteriorBlock(lvl, ib, jb) && loc != kEmpty) {
        for (u32 idx = 0; idx < blockSizeTot; idx++) {
          dataType val = imageData[bIdx*blockSizeTot + idx];
          imageData[bIdx*blockSizeTot + idx] = (val - minVal) / (maxVal - minVal + 1e-16);
          if (f==-1) {
            imageData[bIdx*blockSizeTot + idx] = (val / nLvls);
          }
        }
      }
    }

    // set the pixel values 
    for (uint bIdx=0; bIdx < nBlocks; bIdx++) {
      u64 loc = bLocList[bIdx];
      i32 lvl, ib, jb;
      mortonDecode(loc, lvl, ib, jb);
      if (isInteriorBlock(lvl, ib, jb) && loc != kEmpty) {
        for (uint j = 0; j < blockSize; j++) {
          for (uint i = 0; i < blockSize; i++) {
            u32 idx = i + blockSize * j + bIdx*blockSizeTot;
            u32 nPixels = powi(2,(nLvls - 1 - lvl));
            for (uint jj=0; jj<nPixels; jj++) {
              for (uint ii=0; ii<nPixels; ii++) {
                u32 iPxl = ib*blockSize*nPixels + i*nPixels + ii;
                u32 jPxl = jb*blockSize*nPixels + j*nPixels + jj;
                image[jPxl][iPxl] = imageData[idx] * 65535;

                if (drawGrid && (ii > 0 && jj > 0)) {
                  image[jPxl][iPxl] = 0;
                }
              }
            }
          }
        }
      }
    }

    // output the image to a png file
    char fileName[50];
    if (f >=0) {
      sprintf(fileName, "output/image%02d_%05d.png", f, imageCounter);
    }
    else {
      sprintf(fileName, "output/grid_%05d.png", imageCounter);
    }
    image.write(fileName);
  }
  imageCounter++;
}

void MultiLevelSparseGrid::computeImageData(i32 f) {
  // set image field data 
  if (f >= 0) {
    dataType *Field = getField(f);
    for (u32 bIdx = 0; bIdx < nBlocks; bIdx++) {
      for (u32 idx = 0; idx < blockSizeTot; idx++) {
        imageData[bIdx*blockSizeTot + idx] = Field[bIdx*blockSizeTot + idx];
      }
    }
  }
  else {
    // set grid res level
    for (u32 bIdx = 0; bIdx < nBlocks; bIdx++) {
      u64 loc = bLocList[bIdx];
      i32 lvl, ib, jb;
      mortonDecode(loc, lvl, ib, jb);
      for (u32 idx = 0; idx < blockSizeTot; idx++) {
        u32 flag = cFlagsList[bIdx*blockSizeTot + idx];
        imageData[bIdx*blockSizeTot + idx] = lvl+1 - (2-flag)/2;
      }
    }
  }
}

/*
void MultiLevelSparseGrid::resetBlockCounter(void) {
  zeroBlockCounter<<<1, 1>>>(*this);
}
*/
