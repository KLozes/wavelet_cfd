#include "hip/hip_runtime.h"
#include <thrust/sort.h>
#include <algorithm>

#include <png++/png.hpp>
#include "MultiLevelSparseGrid.cuh"
#include "MultiLevelSparseGridKernels.cuh"

MultiLevelSparseGrid::MultiLevelSparseGrid(real *domainSize_, u32 *baseGridSize_, u32 nLvls_, u32 nFields_) {

  domainSize[0] = domainSize_[0];
  domainSize[1] = domainSize_[1];

  baseGridSize[0] = baseGridSize_[0];
  baseGridSize[1] = baseGridSize_[1];

  nLvls = nLvls_;
  nFields = nFields_;

  imageSize[0] = (baseGridSize[0])*powi(2,nLvls-1);  // image size is the max resolution not including boundary condition blocks
  imageSize[1] = (baseGridSize[1])*powi(2,nLvls-1);

  nBlocks = 0;
  blockCounter = 0;
  imageCounter = 0;

  // grid size checking
  assert(isPowerOf2(blockSize));
  assert(baseGridSize[0]*baseGridSize[1]/blockSize/blockSize < nBlocksMax);

  hipMallocManaged(&bLocList, nBlocksMax*sizeof(u64));
  hipMallocManaged(&bIdxList, nBlocksMax*sizeof(u32));

  hipMallocManaged(&prntIdxList, nBlocksMax*sizeof(u32));
  hipMallocManaged(&prntIdxListOld, nBlocksMax*sizeof(u32));
  hipMallocManaged(&chldIdxList, 4*nBlocksMax*sizeof(u32));
  hipMallocManaged(&chldIdxListOld, 4*nBlocksMax*sizeof(u32));
  hipMallocManaged(&nbrIdxList, 9*nBlocksMax*sizeof(u32));

  hipMallocManaged(&bFlagsList, nBlocksMax*sizeof(u32));
  hipMallocManaged(&cFlagsList, blockSizeTot*nBlocksMax*sizeof(u32));

  hipMallocManaged(&fieldData, nFields*blockSizeTot*nBlocksMax*sizeof(real));
  hipMallocManaged(&imageData, imageSize[0]*imageSize[1]*sizeof(real));

  hipMemset(bLocList, 0, nBlocksMax*sizeof(u64));
  hipMemset(bIdxList, 0, nBlocksMax*sizeof(u32));

  hipMemset(prntIdxList, 0, nBlocksMax*sizeof(u32));
  hipMemset(prntIdxListOld, 0, nBlocksMax*sizeof(u32));
  hipMemset(chldIdxList, 0, 4*nBlocksMax*sizeof(u32));
  hipMemset(chldIdxListOld, 0, 4*nBlocksMax*sizeof(u32));
  hipMemset(nbrIdxList, 0, 9*nBlocksMax*sizeof(u32));

  hipMemset(bFlagsList, 0, nBlocksMax*sizeof(u32));
  hipMemset(cFlagsList, 0, blockSizeTot*nBlocksMax*sizeof(u32));

  hipMemset(fieldData, 0, nFields*blockSizeTot*nBlocksMax*sizeof(real));
  hipMemset(imageData, 0, imageSize[0]*imageSize[1]*sizeof(real));

  hipDeviceSynchronize();
}

MultiLevelSparseGrid::~MultiLevelSparseGrid(void) {
  hipDeviceSynchronize();
  hipFree(bLocList);
  hipFree(bIdxList);
  hipFree(prntIdxList);
  hipFree(nbrIdxList);
  hipFree(cFlagsList);
  hipFree(fieldData);
  hipFree(imageData);
}

void MultiLevelSparseGrid::initializeBaseGrid(void) {

  initTreeKernel<<<nBlocksMax/cudaBlockSize+1, cudaBlockSize>>>(*this);
  hipDeviceSynchronize();

  // fill tree with base grid blocks
  nBlocks = 0;
  for(i32 j=-1; j<baseGridSize[1]/blockSize+1; j++) {
    for (i32 i=-1; i<baseGridSize[0]/blockSize+1; i++) {
      bLocList[nBlocks] = encode(0,i,j);
      bIdxList[nBlocks] = nBlocks;
      nBlocks++; 
    }
  }
}

void MultiLevelSparseGrid::adaptGrid(void) {

  if (nLvls > 1) {
    addFineBlocksKernel<<<1000, cudaBlockSize>>>(*this);
    setBlocksKeepKernel<<<1000, cudaBlockSize>>>(*this);
    //addAdjacentBlocksKernel<<<1000, cudaBlockSize>>>(*this);
    //for(i32 lvl=nLvls-1; lvl>0; lvl--) {
    //  setBlocksKeepKernel<<<1000, cudaBlockSize>>>(*this);
    //  addReconstructionBlocksKernel<<<1000, cudaBlockSize>>>(*this);
    //}
    addBoundaryBlocksKernel<<<1000, cudaBlockSize>>>(*this);
    setBlocksKeepKernel<<<1000, cudaBlockSize>>>(*this);
    deleteDataKernel<<<1000, cudaBlockSize>>>(*this);
  }
}

void MultiLevelSparseGrid::sortBlocks(void) {

  hipDeviceSynchronize();
  //thrust::sort_by_key(thrust::device, bLocList, bLocList+nBlocks, bIdxList);
  //sortFieldData();
  //updateTreeIndicesKernel<<<1000, cudaBlockSize>>>(*this);
  //copyTreeIndicesKernel<<<1000, cudaBlockSize>>>(*this);
  updateNbrIndicesKernel<<<1000, cudaBlockSize>>>(*this);
  flagActiveCellsKernel<<<1000, cudaBlockSize>>>(*this);
  flagParentCellsKernel<<<1000, cudaBlockSize>>>(*this); 
  hipDeviceSynchronize();
}

__device__ void MultiLevelSparseGrid::getCellPos(i32 lvl, i32 ib, i32 jb, i32 i, i32 j, real *pos) {
  pos[0] = (ib*blockSize + i + .5)*getDx(lvl);
  pos[1] = (jb*blockSize + j + .5)*getDy(lvl);
}

__device__ u32 MultiLevelSparseGrid::getNbrIdx(u32 bIdx, i32 i, i32 j) {
  i += blockSize;
  j += blockSize;
  i32 ib = i / blockSize;
  i32 jb = j / blockSize;
  i32 nbrIdx = nbrIdxList[9*bIdx + ib + 3*jb];
  return blockSizeTot*nbrIdx + (i%blockSize) + (j%blockSize)*blockSize;
}

__device__ real MultiLevelSparseGrid::getDx(i32 lvl) {
  return real(domainSize[0])/real(baseGridSize[0]*powi(2,lvl));
}

__device__ real MultiLevelSparseGrid::getDy(i32 lvl) {
  return real(domainSize[1])/real(baseGridSize[1]*powi(2,lvl));
}

__device__ bool MultiLevelSparseGrid::isInteriorBlock(i32 lvl, i32 i, i32 j) { 
  i32 gridSize[2] = {i32(baseGridSize[0]/blockSize*powi(2,lvl)), 
                     i32(baseGridSize[1]/blockSize*powi(2,lvl))};
  return i >= 0 && j >= 0 && i < gridSize[0] && j < gridSize[1];
}

__device__ bool MultiLevelSparseGrid::isExteriorBlock(i32 lvl, i32 i, i32 j) {
  return !isInteriorBlock(lvl, i, j);
}

__host__ __device__ real* MultiLevelSparseGrid::getField(u32 f) {
  return &fieldData[f*nBlocksMax*blockSizeTot];
}

__device__ void MultiLevelSparseGrid::activateBlock(i32 lvl, i32 i, i32 j) {

  u64 loc = encode(lvl, i, j);

  // start at the base grid level accounting for boundary blocks
  i32 iBase = i / powi(2,lvl) + 1; 
  i32 jBase = j / powi(2,lvl) + 1;
  i32 prntIdx = iBase + jBase * (baseGridSize[0]+2);

  for(i32 l = 1; l <= lvl; l++) {
    i32 ib = i / powi(2, lvl-l); // this stuff wrong
    i32 jb = j / powi(2, lvl-l);
    u64 locb = encode(l, ib, jb);
    u32 cIdx = 4*prntIdx + 2*((jb+2)%2) + (ib+2)%2;

    // swap in a temp index if it is empty
    uint prev = atomicCAS(&chldIdxList[cIdx], bEmpty, bEmpty-1);

    // wait until temp index changes to a real index
    while(chldIdxList[cIdx] == bEmpty-1) {
      // if the previous value of the atomicCAS was empty,
      // increment the nBlocks counter create the child block
      if (prev == bEmpty) {
        u32 idx = atomicAdd(&nBlocks, 1);
        bIdxList[idx] = idx;
        bLocList[idx] = locb;
        prntIdxList[idx] = prntIdx;
        chldIdxList[4*idx] = bEmpty;
        chldIdxList[4*idx+1] = bEmpty;
        chldIdxList[4*idx+2] = bEmpty;
        chldIdxList[4*idx+3] = bEmpty;
        bFlagsList[idx] = NEW;
        chldIdxList[cIdx] = idx;
      }
    }
    __threadfence();
    prntIdx = chldIdxList[cIdx];
  }
}

__device__ u32 MultiLevelSparseGrid::getBlockIdx(i32 lvl, i32 i, i32 j) {

  u64 loc = encode(lvl, i, j);

  // search up the tree starting from the base
  i32 iBase = i / powi(2,lvl) + 1; 
  i32 jBase = j / powi(2,lvl) + 1;
  i32 prntIdx = iBase + jBase * (baseGridSize[0]+2);
  
  for(i32 l = 1; l <= lvl; l++) {
    i32 ib = i / powi(2, lvl-l);
    i32 jb = j / powi(2, lvl-l);
    u64 locb = encode(l, ib, jb);
    u32 chldIdx = chldIdxList[4*prntIdx + 2*((jb+2)%2) + (ib+2)%2];
    prntIdx = chldIdx;
    if (prntIdx == bEmpty) {
      break;
    }
  }
  return prntIdx;
}

/*
// seperate bits from a given integer 3 positions apart
__device__ u64 MultiLevelSparseGrid::split(u32 a) {
  u64 x = (u64)a & ((1<<20)-1); // we only look at the first 20 bits
  x = (x | x << 32) & 0x1f00000000ffff;
  x = (x | x << 16) & 0x1f0000ff0000ff;
  x = (x | x << 8) & 0x100f00f00f00f00f;
  x = (x | x << 4) & 0x10c30c30c30c30c3;
  x = (x | x << 2) & 0x1249249249249249;
  return x;
}

// encode ijk indices and resolution level into morton code
__device__ u64 MultiLevelSparseGrid::encode(i32 lvl, i32 i, i32 j) {
  u64 morton = 0;
  i += 1; // add one so that boundary blocks are no longer negative negative
  j += 1;
  morton |= (u64)lvl << 60 | split(i) | split(j) << 1;
  return morton;
}

// compact separated bits into into an integer
__device__ u32 MultiLevelSparseGrid::compact(u64 w) {
  w &=                  0x1249249249249249;
  w = (w ^ (w >> 2))  & 0x30c30c30c30c30c3;
  w = (w ^ (w >> 4))  & 0xf00f00f00f00f00f;
  w = (w ^ (w >> 8))  & 0x00ff0000ff0000ff;
  w = (w ^ (w >> 16)) & 0x00ff00000000ffff;
  w = (w ^ (w >> 32)) & 0x00000000001fffff;
  return (u32)w;
}

// decode morton code into ij idx and resolution level
__device__ void MultiLevelSparseGrid::decode(u64 morton, i32 &lvl, i32 &i, i32 &j) {
  lvl = i32((morton & ((u64)15 << 60)) >> 60);   // get the level stored in the last 4 bits
  morton &= ~ ((u64)15 << 60); // remove the last 4 bits
  i = compact(morton) - 1; 
  j = compact(morton >> 1) - 1;
}

*/

// encode ijk indices and resolution level into locational code
__device__ u64 MultiLevelSparseGrid::encode(i32 lvl, i32 i, i32 j) {
  i += 1; // add one so that boundary blocks are no longer negative negative
  j += 1;
  u64 loc = 0;
  loc |= (u64)lvl << 60 | (u64)j << 20 | (u64)i;
  return loc;
}

// decode locational code into ij idx and resolution level
__device__ void MultiLevelSparseGrid::decode(u64 loc, i32 &lvl, i32 &i, i32 &j) {
  lvl = loc >> 60;
  j = ((loc >> 20) & ((1 << 20)-1)) - 1;
  i = (loc & ((1 << 20)-1)) - 1;
}

void MultiLevelSparseGrid::paint(void) {

  hipDeviceSynchronize();
  png::image<png::gray_pixel_16> image(imageSize[0], imageSize[1]);

  for (i32 f=-1; f<4; f++) {
    //computeImageData(f);
    computeImageDataKernel<<<1000, cudaBlockSize>>>(*this, f);
    hipDeviceSynchronize();

    // normalize image data and fill png image
    real maxVal = -1e32;
    real minVal = 1e32;

    for (i32 idx=0; idx<imageSize[0]*imageSize[1]; idx++) {
      maxVal = fmax(maxVal, imageData[idx]);
      minVal = fmin(minVal, imageData[idx]);
    }

    if (f == -1) {
      minVal = 0;
      maxVal = nLvls;
    }
 
    for (i32 j=0; j<imageSize[1]; j++) {
      for (i32 i=0; i<imageSize[0]; i++) {
        i32 idx = j*imageSize[1] + i;
        image[j][i] = (imageData[idx] - minVal) / (maxVal - minVal + 1e-16) * 65535;
      }
    }

    // output the image to a png file
    char fileName[50];
    if (f >=0) {
      sprintf(fileName, "output/image%02d_%05d.png", f, imageCounter);
    }
    else {
      sprintf(fileName, "output/grid_%05d.png", imageCounter);
    }
    image.write(fileName);
  }
  imageCounter++;
}

void MultiLevelSparseGrid::computeImageData(i32 f) {

  real *U;
  if (f >= 0) {
    U = getField(f);
  }

  bool gridOn = true;

  // set the pixel values 
  for (uint bIdx=0; bIdx < nBlocks; bIdx++) {
    u64 loc = bLocList[bIdx];
    i32 lvl, ib, jb;
    decode(loc, lvl, ib, jb);
    if (isInteriorBlock(lvl, ib, jb) && loc != kEmpty) {
      for (uint j = 0; j < blockSize; j++) {
        for (uint i = 0; i < blockSize; i++) {
          u32 idx = i + blockSize * j + bIdx*blockSizeTot;
          u32 nPixels = powi(2,(nLvls - 1 - lvl));
          for (uint jj=0; jj<nPixels; jj++) {
            for (uint ii=0; ii<nPixels; ii++) {
              u32 iPxl = ib*blockSize*nPixels + i*nPixels + ii;
              u32 jPxl = jb*blockSize*nPixels + j*nPixels + jj;
              if (f >= 0) {
                imageData[jPxl*imageSize[0] + iPxl] = U[idx];
              }
              else {
                u32 cFlag = cFlagsList[idx];
                imageData[jPxl*imageSize[0] + iPxl] = lvl+1 - (2-cFlag)/2;
              }
              if (gridOn && ii > 0 && jj > 0) {
                  imageData[jPxl*imageSize[0] + iPxl] = 0;
              }
            }
          }
        }
      }
    }
  }
}

/*
void MultiLevelSparseGrid::resetBlockCounter(void) {
  zeroBlockCounter<<<1000, cudaBlockSize>>>(*this);
}
*/
